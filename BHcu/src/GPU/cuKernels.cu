#include "hip/hip_runtime.h"
/*--------------------------------*- BHcu -*-----------------*---------------*\
| #####   ##  ##                |                            | Version 1.0    |
| ##  ##  ##  ##   ####  ##  ## |  BHcu: Barnes-Hut method   | 2021/08/05     |
| #####   ######  ##     ##  ## |  for 2D vortex particles   *----------------*
| ##  ##  ##  ##  ##     ##  ## |  Open Source Code                           |
| #####   ##  ##   ####   ####  |  https://www.github.com/vortexmethods/fastm |
|                                                                             |
| Copyright (C) 2020-2021 Ilia Marchevsky, Evgeniya Ryatina                   |
| Copyright (C) 2013, Texas State University-San Marcos. All rights reserved. |
*-----------------------------------------------------------------------------*
| File name: cuKernels.cu                                                     |
| Info: Source code of BHcu                                                   |
|                                                                             |
| This file is part of BHcu.                                                  |
| BHcu is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| BHcu is distributed in the hope that it will be useful, but WITHOUT         |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with BHcu.  If not, see <http://www.gnu.org/licenses/>.               |
\*---------------------------------------------------------------------------*/

/*!
\file
\brief Реализация CUDA-ядер
\author Марчевский Илья Константинович
\author Рятина Евгения Павловна
\version 1.0
\date 05 августа 2021 г.
*/



#include "cuKernels.cuh"

#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>


#include "operations.cuh"
#include "Point2D.h"

#define WARPSIZE 32
#define MAXDEPTH 32
#define BLOCKD 32


namespace BHcu
{

int blocks;
__device__ volatile int stepd, bottomd, maxdepthd;
__device__ unsigned int blkcntd;
__device__ volatile real radiusd;



void setBlocks(int& blocks_)
{
     blocks_ = blocks;
}

void CudaSelect(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "There is no device supporting CUDA\n");
        exit(-1);
    }

    if ((dev < 0) || (deviceCount <= dev)) {
        fprintf(stderr, "There is no device %d\n", dev);
        exit(-1);
    }
    hipSetDevice(dev);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
        fprintf(stderr, "There is no CUDA capable device\n");
        exit(-1);
    }
    if (deviceProp.major < 2) {
        fprintf(stderr, "Need at least compute capability 2.0\n");
        exit(-1);
    }
    if (deviceProp.warpSize != WARPSIZE) {
        fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
        exit(-1);
    }

    blocks = deviceProp.multiProcessorCount;
    
    if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE - 1) != 0)) {
        fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
        exit(-1);
    }
    if (MAXDEPTH > WARPSIZE) {
        fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
        exit(-1);
    }
    if ((THREADS1 <= 0) || (THREADS1 & (THREADS1 - 1) != 0)) {
        fprintf(stderr, "THREADS1 must be greater than zero and a power of two\n");
        exit(-1);
    }

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);

    int fact = 1024;
    int driverVersion, runtimeVersion;

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("\n");
    printf("                          GPU Device Properties                         \n");
    printf("------------------------------------------------------------------------\n");
    printf("Name:                                  %s\n", properties.name); 
    printf("CUDA driver/runtime version:           %d.%d/%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("CUDA compute capabilitiy:              %d.%d\n", properties.major, properties.minor);
    printf("Number of multiprocessors:             %d\n", properties.multiProcessorCount);
    if (printFullCUDAinfo)
    {
        printf("GPU clock rate:                        %d (MHz)\n", properties.clockRate / fact);
        printf("Memory clock rate:                     %d (MHz)\n", properties.memoryClockRate / fact);
        printf("Memory bus width:                      %d-bit\n", properties.memoryBusWidth);
        printf("Theoretical memory bandwidth:          %d (GB/s)\n", (properties.memoryClockRate / fact * (properties.memoryBusWidth / 8) * 2) / fact);
        printf("Device global memory:                  %d (MB)\n", (int)(properties.totalGlobalMem / (fact * fact)));
        printf("Shared memory per block:               %d (KB)\n", (int)(properties.sharedMemPerBlock / fact));
        printf("Constant memory:                       %d (KB)\n", (int)(properties.totalConstMem / fact));
        printf("Maximum number of threads per block:   %d\n", properties.maxThreadsPerBlock);
        printf("Maximum thread dimension:              [%d, %d, %d]\n", properties.maxThreadsDim[0], properties.maxThreadsDim[1], properties.maxThreadsDim[2]);
        printf("Maximum grid size:                     [%d, %d, %d]\n", properties.maxGridSize[0], properties.maxGridSize[1], properties.maxGridSize[2]);
    }
    printf("------------------------------------------------------------------------\n");  
}


void cudaDelete(void* cudaPtr)
{
	hipFree(cudaPtr);
}


void* cudaNew(int n, size_t sizeType)
{
	void* cudaPtr;
	hipMalloc(&cudaPtr, sizeType * n);
	CudaTest("couldn't allocate device memory");

	return cudaPtr;
}

void cudaCopyVecToDevice(void* hostPtr, void* cudaPtr, size_t n, size_t typeSize)
{
	hipMemcpy(cudaPtr, hostPtr, typeSize * n, hipMemcpyHostToDevice);
	CudaTest("couldn't copy data from host to device");
}

void cudaCopyVecFromDevice(void* cudaPtr, void* hostPtr, size_t n, size_t typeSize)
{
	hipMemcpy(hostPtr, cudaPtr, typeSize * n, hipMemcpyDeviceToHost);
	CudaTest("couldn't copy data from device to host");
}


//////////////////
/// Error TEST
//////////////////


void CudaTest(const char* msg)
{
    hipError_t e;

    //hipDeviceSynchronize();
    hipDeviceSynchronize();
    if (hipSuccess != (e = hipGetLastError())) {
        fprintf(stderr, "%s: %d\n", msg, e);
        fprintf(stderr, "%s\n", hipGetErrorString(e));
        exit(-1);
    }
}


//////////////////
/// CUDA Kernels
//////////////////


/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void InitializationKernel(int* __restrict errd)
{
    *errd = 0;
    stepd = -1;
    maxdepthd = 1;
    blkcntd = 0;
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS1, FACTOR1)
void BoundingBoxKernel(int nnodesd, int nbodiesd, volatile int* __restrict startd, volatile int* __restrict childd, volatile int* __restrict massd, volatile moms* __restrict momd, volatile real2* __restrict posd, volatile real2* __restrict maxrd, volatile real2* __restrict minrd)
{
    register int i, j, k, inc;
    register real2 val;
    register real2 minr, maxr;
    __shared__ volatile real2 sminr[THREADS1], smaxr[THREADS1];

    // initialize with valid data (in case #bodies < #threads)
    minr.x = maxr.x = posd[0].x;
    minr.y = maxr.y = posd[0].y;

    // scan all bodies
    i = threadIdx.x;
    inc = THREADS1 * gridDim.x;
    for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
        val.x = posd[j].x;
        val.y = posd[j].y;

        minr.x = realmin(minr.x, val.x);
        maxr.x = realmax(maxr.x, val.x);

        minr.y = realmin(minr.y, val.y);
        maxr.y = realmax(maxr.y, val.y);
    }

    // reduction in shared memory
    sminr[i].x = minr.x;
    smaxr[i].x = maxr.x;
    sminr[i].y = minr.y;
    smaxr[i].y = maxr.y;

    for (j = THREADS1 / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sminr[i].x = minr.x = realmin(minr.x, sminr[k].x);
            smaxr[i].x = maxr.x = realmax(maxr.x, smaxr[k].x);
            sminr[i].y = minr.y = realmin(minr.y, sminr[k].y);
            smaxr[i].y = maxr.y = realmax(maxr.y, smaxr[k].y);
        }
    }

    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        minrd[k].x = minr.x;
        maxrd[k].x = maxr.x;
        minrd[k].y = minr.y;
        maxrd[k].y = maxr.y;

        

        __threadfence();

        inc = gridDim.x - 1;
        if (inc == atomicInc(&blkcntd, inc)) {
            // I'm the last block, so combine all block results
            for (j = 0; j <= inc; j++) {
                minr.x = realmin(minr.x, minrd[j].x);
                maxr.x = realmax(maxr.x, maxrd[j].x);
                minr.y = realmin(minr.y, minrd[j].y);
                maxr.y = realmax(maxr.y, maxrd[j].y);
            }

            // compute 'radius'
            radiusd = realmax(maxr.x - minr.x, maxr.y - minr.y) / 2;

            // create root node
            k = nnodesd;
            bottomd = k;

            massd[k] = -1;
            momd[k].gam = 0;
#ifdef USE_DIP
            momd[k].dip.x = 0;
            momd[k].dip.y = 0;
#ifdef USE_QUA
            momd[k].qua.x = 0;
            momd[k].qua.y = 0;
#ifdef USE_OCT
            momd[k].oct.x = 0;
            momd[k].oct.y = 0;
#ifdef USE_HEX            
            momd[k].hex.x = 0;
            momd[k].hex.y = 0;
#endif
#endif
#endif
#endif
            startd[k] = 0;
            posd[k].x = (minr.x + maxr.x) / 2;
            posd[k].y = (minr.y + maxr.y) / 2;
            k *= 4;
            for (i = 0; i < 4; i++) childd[k + i] = -1;

            stepd++;
        }
    }
}





/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(1024, 1)
void ClearKernel1(int nnodesd, int nbodiesd, volatile int* __restrict childd)
{
    register int k, inc, top, bottom;

    top = 4 * nnodesd;
    bottom = 4 * nbodiesd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    if (k < bottom) k += inc;

    // iterate over all cells assigned to thread
    while (k < top) {
        childd[k] = -1;
        k += inc;
    }
}


__global__
__launch_bounds__(THREADS2, FACTOR2)
void TreeBuildingKernel(int nnodesd, int nbodiesd, volatile int* __restrict errd, volatile int* __restrict childd, volatile real2* __restrict posd)
{
    register int i, j, depth, localmaxdepth, skip, inc;
    register real x, y, r;
    register real2 p;
    register real2 dr;
    register int ch, n, cell, locked, patch;
    register real radius;
    register real2 root;

    // cache root data
    radius = radiusd;

	//if ((threadIdx.x == 0) && (blockIdx.x == 0))
	//	printf("radius = %f\n", radius);
	//	printf("pod[10].g = %f, %f\n", posd[10].x, posd[10].y);
	//	printf("nn = %d, %d\n", nnodesd, nbodiesd);

    root.x = posd[nnodesd].x;
    root.y = posd[nnodesd].y;

    localmaxdepth = 1;
    skip = 1;
    inc = blockDim.x * gridDim.x;
    i = threadIdx.x + blockIdx.x * blockDim.x;

    // iterate over all bodies assigned to thread
    while (i < nbodiesd) {
        p.x = posd[i].x;
        p.y = posd[i].y;

        if (skip != 0) {
            // new body, so start traversing at root
            skip = 0;
            p.x = posd[i].x;
            p.y = posd[i].y;

            n = nnodesd;
            depth = 1;
            r = radius / 2;
            dr.x = dr.y = -r;
            j = 0;
            // determine which child to follow
            if (root.x < p.x) { j = 1; dr.x = r; }
            if (root.y < p.y) { j |= 2; dr.y = r; }
            x = root.x + dr.x;
            y = root.y + dr.y;
        }

        // follow path to leaf cell
        ch = childd[n * 4 + j];
        while (ch >= nbodiesd) {
            n = ch;
            depth++;
            r /= 2;
            dr.x = dr.y = -r;
            j = 0;
            // determine which child to follow
            if (x < p.x) { j = 1; dr.x = r; }
            if (y < p.y) { j |= 2; dr.y = r; }
            x += dr.x;
            y += dr.y;
            ch = childd[n * 4 + j];
        }

        if (ch != -2) {  // skip if child pointer is locked and try again later
            locked = n * 4 + j;
            if (ch == -1) {
                if (-1 == atomicCAS((int*)&childd[locked], -1, i)) {  // if null, just insert the new body
                    localmaxdepth = max(depth, localmaxdepth);
                    i += inc;  // move on to next body
                    skip = 1;
                }
            }
            else {  // there already is a body in this position
                if (ch == atomicCAS((int*)&childd[locked], ch, -2)) {  // try to lock
                    patch = -1;
                    // create new cell(s) and insert the old and new body
                    do {
                        depth++;

                        cell = atomicSub((int*)&bottomd, 1) - 1;
                        if (cell <= nbodiesd) {
                            *errd = 1;
							//printf("!!!");
                            bottomd = nnodesd;
                        }

                        if (patch != -1) {
                            childd[n * 4 + j] = cell;
                        }
                        patch = max(patch, cell);

                        j = 0;
                        if (x < posd[ch].x) j = 1;
                        if (y < posd[ch].y) j |= 2;
                        childd[cell * 4 + j] = ch;

                        posd[cell].x = x;
                        posd[cell].y = y;


                        n = cell;
                        r /= 2;
                        dr.x = dr.y = -r;
                        j = 0;
                        if (x < p.x) { j = 1; dr.x = r; }
                        if (y < p.y) { j |= 2; dr.y = r; }
                        x += dr.x;
                        y += dr.y;

                        ch = childd[n * 4 + j];
                        // repeat until the two bodies are different children
                    } while (ch >= 0);
                    childd[n * 4 + j] = i;

                    localmaxdepth = max(depth, localmaxdepth);
                    i += inc;  // move on to next body
                    skip = 2;
                }
            }
        }
        __syncthreads();
        __threadfence();

        if (skip == 2) {
            childd[locked] = patch;
        }
    }
    // record maximum tree depth
    atomicMax((int*)&maxdepthd, localmaxdepth);
}


__global__
__launch_bounds__(1024, 1)
void ClearKernel2(int nnodesd, volatile int* __restrict startd, volatile int* __restrict massd)
{
    register int k, inc, bottom;

    bottom = bottomd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    if (k < bottom) k += inc;

    // iterate over all cells assigned to thread
    while (k < nnodesd) {
        massd[k] = -1;
        startd[k] = -1;
        k += inc;
    }
}


__global__
__launch_bounds__(1024, 1)
void ClearKernel3(int nnodesd, int nbodies, const real* __restrict gamd, volatile moms* __restrict momd)
{
    register int k, inc;

    inc = blockDim.x * gridDim.x;
    k = threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    
    // iterate over all cells assigned to thread
    while (k < nnodesd) {        
        momd[k].gam = (k < nbodies) ? gamd[k] : 0;
#ifdef USE_DIP
        momd[k].dip.x = 0;
        momd[k].dip.y = 0;
#ifdef USE_QUA
        momd[k].qua.x = 0;
        momd[k].qua.y = 0;
#ifdef USE_OCT
        momd[k].oct.x = 0;
        momd[k].oct.y = 0;
#ifdef USE_HEX
        momd[k].hex.x = 0;
        momd[k].hex.y = 0;
#endif
#endif
#endif
#endif        
        k += inc;
    }
}





/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel(const int nnodesd, const int nbodiesd, volatile int* __restrict countd, const int* __restrict childd, volatile int* __restrict massd, volatile moms* __restrict momd, volatile real2* __restrict posd)
{
    register int i, j, k, ch, inc, cnt, bottom, flag;
    register real g, cg;
#ifdef USE_DIP
    register real2 cen, dr;
    register real2 d, cd, mh;
#ifdef USE_QUA
    register real2 q, cq, dh;
#ifdef USE_OCT
    register real2 o, co, qh;
#ifdef USE_HEX
    register real2 h, chex, oh;
#endif
#endif
#endif
#endif
    register int m, cm;
    __shared__ int child[THREADS3 * 4];


    bottom = bottomd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    if (k < bottom) k += inc;

    register int restart = k;
    for (j = 0; j < 5; j++) {  // wait-free pre-passes
      // iterate over all cells assigned to thread
        while (k <= nnodesd) {
            if (massd[k] < 0) {
                for (i = 0; i < 4; i++) {
                    ch = childd[k * 4 + i];
                    child[i * THREADS3 + threadIdx.x] = ch;  // cache children 
                    if ((ch >= nbodiesd) && (massd[ch] < 0)) {
                        break;
                    }
                }

                if (i == 4) {
                    // all children are ready
                    cm = 0;
                    cg = 0;
                    cnt = 0;
#ifdef USE_DIP
                    cen.x = posd[k].x;
                    cen.y = posd[k].y;
                    cd.x = cd.y = 0;
#ifdef USE_QUA
                    cq.x = cq.y = 0;
#ifdef USE_OCT
                    co.x = co.y = 0;
#ifdef USE_HEX
                    chex.x = chex.y = 0;
#endif
#endif
#endif
#endif
                    for (i = 0; i < 4; i++) {
                        //ch = childd[k * 4 + i];
                        ch = child[i * THREADS3 + threadIdx.x];
                        if (ch >= 0) {
                            g = momd[ch].gam;
#ifdef USE_DIP
                            dr.x = posd[ch].x - cen.x; dr.y = posd[ch].y - cen.y;
                            d.x = momd[ch].dip.x; d.y = momd[ch].dip.y;
#ifdef USE_QUA

                            q.x = momd[ch].qua.x; q.y = momd[ch].qua.y;
#ifdef USE_OCT

                            o.x = momd[ch].oct.x; o.y = momd[ch].oct.y;
#ifdef USE_HEX

                            h.x = momd[ch].hex.x; h.y = momd[ch].hex.y;
#endif
#endif
#endif
#endif
                            m = massd[ch];

                            cnt += (ch >= nbodiesd) ? countd[ch] : 1;

                            // add child's contribution                                                        
                            cg += g;
#ifdef USE_DIP
                            mh = g * dr;
                            cd += d + mh;
#ifdef USE_QUA
                            mh = multz(mh, dr);
                            dh = multz(d, dr);
                            cq += q + 2 * dh + mh;
#ifdef USE_OCT
                            mh = multz(mh, dr);
                            dh = multz(dh, dr);
                            qh = multz(q, dr);
                            co += o + 3 * qh + 3 * dh + mh;
#ifdef USE_HEX
                            mh = multz(mh, dr);
                            dh = multz(dh, dr);
                            qh = multz(qh, dr);
                            oh = multz(o, dr);
                            chex += h + 4 * oh + 6 * qh + 4 * dh + mh;
#endif
#endif
#endif
#endif

                            cm += m;
                        }
                    }
                    countd[k] = cnt;

                    momd[k].gam = cg;
#ifdef USE_DIP
                    momd[k].dip.x = cd.x;
                    momd[k].dip.y = cd.y;
#ifdef USE_QUA
                    momd[k].qua.x = cq.x;
                    momd[k].qua.y = cq.y;
#ifdef USE_OCT
                    momd[k].oct.x = co.x;
                    momd[k].oct.y = co.y;
#ifdef USE_HEX
                    momd[k].hex.x = chex.x;
                    momd[k].hex.y = chex.y;
#endif
#endif
#endif
#endif

                    __threadfence();  // make sure data are visible before setting mass

                    massd[k] = cm;
                }
            }
            k += inc;  // move on to next cell
        }
        k = restart;
    }

    flag = 0;
    j = 0;
    // iterate over all cells assigned to thread
    while (k <= nnodesd) {
        if (massd[k] >= 0) {
            k += inc;
        }
        else {
            if (j == 0) {
                j = 4;
                for (i = 0; i < 4; i++) {
                    ch = childd[k * 4 + i];
                    child[i * THREADS3 + threadIdx.x] = ch;  // cache children                                 

                    if ((ch < nbodiesd) || (massd[ch] >= 0)) {
                        j--;
                    }

                }
            }
            else {
                j = 4;
                for (i = 0; i < 4; i++) {
                    //ch = childd[k * 4 + i];
                    ch = child[i * THREADS3 + threadIdx.x];

                    if ((ch < nbodiesd) || (massd[ch] >= 0))
                    {
                        j--;
                    }
                }
            }

            if (j == 0) {
                // all children are ready
                cg = 0;
                cm = 0;
#ifdef USE_DIP
                cen.x = posd[k].x;
                cen.y = posd[k].y;
                cd.x = cd.y = 0;
#ifdef USE_QUA
                cq.x = cq.y = 0;
#ifdef USE_OCT
                co.x = co.y = 0;
#ifdef USE_HEX
                chex.x = chex.y = 0;
#endif
#endif
#endif
#endif

                cnt = 0;
                for (i = 0; i < 4; i++) {
                    //ch = childd[k * 4 + i];
                    ch = child[i * THREADS3 + threadIdx.x];

                    if (ch >= 0) {
                        g = momd[ch].gam;
#ifdef USE_DIP
                        dr.x = posd[ch].x - cen.x; dr.y = posd[ch].y - cen.y;
                        d.x = momd[ch].dip.x; d.y = momd[ch].dip.y;
#ifdef USE_QUA

                        q.x = momd[ch].qua.x; q.y = momd[ch].qua.y;
#ifdef USE_OCT

                        o.x = momd[ch].oct.x; o.y = momd[ch].oct.y;
#ifdef USE_HEX

                        h.x = momd[ch].hex.x; h.y = momd[ch].hex.y;
#endif
#endif
#endif
#endif
                        m = massd[ch];

                        cnt += (ch >= nbodiesd) ? countd[ch] : 1;

                        // add child's contribution                                                        
                        cg += g;
#ifdef USE_DIP
                        mh = g * dr;
                        cd += d + mh;
#ifdef USE_QUA
                        mh = multz(mh, dr);
                        dh = multz(d, dr);
                        cq += q + 2 * dh + mh;
#ifdef USE_OCT
                        mh = multz(mh, dr);
                        dh = multz(dh, dr);
                        qh = multz(q, dr);
                        co += o + 3 * qh + 3 * dh + mh;
#ifdef USE_HEX
                        mh = multz(mh, dr);
                        dh = multz(dh, dr);
                        qh = multz(qh, dr);
                        oh = multz(o, dr);
                        chex += h + 4 * oh + 6 * qh + 4 * dh + mh;
#endif
#endif
#endif
#endif
                        cm += m;
                    }
                }
                countd[k] = cnt;

                momd[k].gam = cg;
#ifdef USE_DIP
                momd[k].dip.x = cd.x;
                momd[k].dip.y = cd.y;
#ifdef USE_QUA
                momd[k].qua.x = cq.x;
                momd[k].qua.y = cq.y;
#ifdef USE_OCT
                momd[k].oct.x = co.x;
                momd[k].oct.y = co.y;
#ifdef USE_HEX
                momd[k].hex.x = chex.x;
                momd[k].hex.y = chex.y;
#endif
#endif
#endif
#endif
                flag = 1;
            }
        }

        __threadfence();
        __syncthreads();

        if (flag != 0) {

            //atomicExch((real*)&massd[k], cm);
            massd[k] = cm;
            k += inc;
            flag = 0;
        }
    }
}





/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS4, FACTOR4)
void SortKernel(int nnodesd, int nbodiesd, volatile int* __restrict sortd, const int* __restrict countd, volatile int* __restrict startd, volatile int* __restrict childd)
{
    register int i, j, k, ch, dec, start, bottom;

    bottom = bottomd;
    dec = blockDim.x * gridDim.x;
    k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

    // iterate over all cells assigned to thread
    while (k >= bottom) {
        start = startd[k];
        if (start >= 0) {
            j = 0;
            for (i = 0; i < 4; i++) {
                ch = childd[k * 4 + i];
                if (ch >= 0) {
                    if (i != j) {
                        // move children to front (needed later for speed)
                        childd[k * 4 + i] = -1;
                        childd[k * 4 + j] = ch;
                    }
                    j++;
                    if (ch >= nbodiesd) {
                        // child is a cell
                        startd[ch] = start;  // set start ID of child
                        start += countd[ch];  // add #bodies in subtree
                    }
                    else {
                        // child is a body
                        sortd[start] = ch;  // record body in 'sorted' array
                        start++;
                    }
                }
            }
            k -= dec;  // move on to next cell
        }
    }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/


__global__
__launch_bounds__(THREADS5, FACTOR5)
void ForceCalculationKernel(int nnodesd, int nbodiesd, int* __restrict errd, real itolsqd, real epssqd, const int* __restrict sortd,
    const int* __restrict childd, const moms* __restrict momd,
    const real2* __restrict posd, volatile real2* __restrict veld)

{
    register int i, j, k, n, depth, base, sbase, diff, pd, nd;
    register real2 p, v, dr;
    register real r2;
    register moms mom;
#ifdef USE_DIP
    register real2 th;
#endif
    __shared__ volatile int pos[MAXDEPTH * THREADS5 / WARPSIZE], node[MAXDEPTH * THREADS5 / WARPSIZE];
    __shared__ real dq[MAXDEPTH * THREADS5 / WARPSIZE];

    if (0 == threadIdx.x) {
        r2 = radiusd * 2;
        // precompute values that depend only on tree level
        dq[0] = r2 * r2;
        for (i = 1; i < maxdepthd; i++) {
            dq[i] = dq[i - 1] / 4;
            //dq[i - 1];// += epssqd;
        }
        //dq[i - 1];// += epssqd;

        if (maxdepthd > MAXDEPTH) {
            *errd = maxdepthd;
        }
    }
    __syncthreads();

    if (maxdepthd <= MAXDEPTH) {
        // figure out first thread in each warp (lane 0)
        base = threadIdx.x / WARPSIZE;
        sbase = base * WARPSIZE;
        j = base * MAXDEPTH;
        diff = threadIdx.x - sbase;
        // make multiple copies to avoid index calculations later
        if (diff < MAXDEPTH) {
            dq[diff + j] = dq[diff];
        }
        __syncthreads();
        __threadfence_block();

        // iterate over all bodies assigned to thread
        for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
            i = sortd[k];  // get permuted/sorted index
            // cache position info
            p.x = posd[i].x;
            p.y = posd[i].y;

            v.x = 0;
            v.y = 0;

            // initialize iteration stack, i.e., push root node onto stack
            depth = j;
            if (sbase == threadIdx.x) {
                pos[j] = 0;
                node[j] = nnodesd * 4;
            }

            do {
                // stack is not empty
                pd = pos[depth];
                nd = node[depth];
                while (pd < 4) {
                    // node on top of stack has more children to process
                    n = childd[nd + pd];  // load child pointer
                    pd++;

                    if (n >= 0) {
                        dr.x = p.x - posd[n].x;
                        dr.y = p.y - posd[n].y;
                        mom = momd[n];

                        r2 = dr.x * dr.x + dr.y * dr.y;   // compute distance squared (plus softening)
                        if ((n < nbodiesd) || __all_sync(0xffffffff, (dq[depth] + epssqd) * itolsqd < r2)) {  // check if all threads agree that cell is far enough away (or is a body)
                            real f = mom.gam / realmax(r2, epssqd);

                            v += f * dr;
#ifdef USE_DIP 
                            real2 cftr = ((r2 > 0) ? 1 / r2 : 0) * dr;

                            th = multz(cftr, cftr);
                            v += multzA(th, mom.dip);
#ifdef USE_QUA 
                            th = multz(th, cftr);
                            v += multzA(th, mom.qua);
#ifdef USE_OCT 
                            th = multz(th, cftr);
                            v += multzA(th, mom.oct);
#ifdef USE_HEX 
                            th = multz(th, cftr);
                            v += multzA(th, mom.hex);
#endif
#endif
#endif
#endif

                        }
                        else {
                            // push cell onto stack
                            if (sbase == threadIdx.x) {  // maybe don't push and inc if last child
                                pos[depth] = pd;
                                node[depth] = nd;
                            }
                            depth++;
                            pd = 0;
                            nd = n * 4;
                        }
                    }
                    else {
                        pd = 4;  // early out because all remaining children are also zero
                    }
                }
                depth--;  // done with this level
            } while (depth >= j);


            // update velocity
            veld[i].x = -v.y;
            veld[i].y = v.x;
        }
    }
}


/******************************************************************************/
/*** compute force (direct) ***************************************************/
/******************************************************************************/


__global__
//__launch_bounds__(THREADSD, FACTORD)
void ForceDirectCalculationKernel(int nnodesd, int nbodiesd,
    int* __restrict errd,
    real itolsqd, real epssqd,
    const int* __restrict sortd, const int* __restrict childd,
    const moms* __restrict momd,
    const real2* __restrict posd,
    volatile real2* __restrict veld)
{
    //*
    __shared__ real2 shr[BLOCKD];
    __shared__ real shg[BLOCKD];

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    real2 pt;
    pt.x = posd[i].x;
    pt.y = posd[i].y;

    real2 vel;
    vel.x = vel.y = 0;

    real2 dr;
    real dr2, izn;


    //vortices
    for (size_t j = 0; j < nbodiesd; j += BLOCKD)
    {
        shr[threadIdx.x].x = posd[(j + threadIdx.x)].x;
        shr[threadIdx.x].y = posd[(j + threadIdx.x)].y;
        shg[threadIdx.x] = momd[(j + threadIdx.x)].gam;

        __syncthreads();

        for (size_t q = 0; q < BLOCKD; ++q)
        {
            if (j + q < nbodiesd)
            {
                dr.x = pt.x - shr[q].x;
                dr.y = pt.y - shr[q].y;
                dr2 = dr.x * dr.x + dr.y * dr.y;

                izn = shg[q] / realmax(dr2, epssqd);// / CUboundDenom(dr2, eps2); //РЎРіР»Р°Р¶РёРІР°С‚СЊ РЅР°РґРѕ!!!

                vel.x -= dr.y * izn;
                vel.y += dr.x * izn;

            }
        }
        __syncthreads();
    }

    if (i < nbodiesd)
    {
        veld[i].x = vel.x;// * iDPI;
        veld[i].y = vel.y;// * iDPI;
    }
    //*/
}


/******************************************************************************/


void KernelsOptimization()
{
    // set L1/shared memory configuration
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(BoundingBoxKernel), hipFuncCachePreferShared); //1
    hipGetLastError();  // reset error value
    
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(TreeBuildingKernel), hipFuncCachePreferL1);    //2
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel1), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel3), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel), hipFuncCachePreferL1);   //3
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SortKernel), hipFuncCachePreferL1);            //4
    hipGetLastError();  // reset error value
    
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel), hipFuncCachePreferEqual); //5
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceDirectCalculationKernel), hipFuncCachePreferEqual); //d
    hipGetLastError();  // reset error value
}


/******************************************************************************/





//////////////////
/// Wrappers
//////////////////



    /******************************************************************************/
    /*** initialize memory ********************************************************/
    /******************************************************************************/

    float cuInitializationKernel(int* __restrict errd)
    {
        //fprintf(stderr, "IKKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);

        hipEventRecord(start, 0);
        InitializationKernel<<<1, 1>>> (errd);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        CudaTest("kernel 0 launch failed");
        
        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }


    /******************************************************************************/
    /*** compute center and radius ************************************************/
    /******************************************************************************/

    float cuBoundingBoxKernel(
        int nnodesd, int nbodiesd,
        volatile int* __restrict startd, volatile int* __restrict childd,
        volatile int* __restrict massd,
        volatile moms * __restrict momd,
        volatile realPoint* __restrict posd,
        volatile realPoint* __restrict maxrd, volatile realPoint* __restrict minrd)
    {
        //fprintf(stderr, "BBKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        BoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>> (nnodesd, nbodiesd, startd, childd, massd, momd, (real2*)posd, (real2*)maxrd, (real2*)minrd);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 1 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** build tree ***************************************************************/
    /******************************************************************************/

    float cuClearKernel1(int nnodesd, int nbodiesd, volatile int* __restrict childd)
    {
        //fprintf(stderr, "C1Kernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);        

        ClearKernel1 << <blocks * 1, 1024 >> > (nnodesd, nbodiesd, childd);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel clear1 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    float cuTreeBuildingKernel(
        int nnodesd, int nbodiesd,
        volatile int* __restrict errd, volatile int* __restrict childd,
        volatile realPoint* __restrict posd)
    {
        //fprintf(stderr, "TBKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        TreeBuildingKernel << <blocks * FACTOR2, THREADS2 >> > (nnodesd, nbodiesd, errd, childd, (real2*)posd);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);        
        return time;
    }

    float cuClearKernel23(
        int nnodesd, int nbodiesd,
        volatile int* __restrict startd,
        volatile int* __restrict massd,
        const real* __restrict gamd,
        volatile moms* __restrict momd)
    {
        //fprintf(stderr, "CxKernel\n");
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        ClearKernel2 << <blocks * 1, 1024 >> > (nnodesd, startd, massd);
        ClearKernel3 << <blocks * 1, 1024 >> > (nnodesd, nbodiesd, gamd, momd);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel clear2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** compute multipole moments for all the cells ******************************/
    /******************************************************************************/

    float cuSummarizationKernel(
        const int nnodesd, const int nbodiesd,
        volatile int* __restrict countd, const int* __restrict childd,
        volatile int* __restrict massd,
        volatile moms * __restrict momd,
        volatile realPoint* __restrict posd)
    {
        //fprintf(stderr, "SKKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        SummarizationKernel << <blocks * FACTOR3, THREADS3 >> > (nnodesd, nbodiesd, countd, childd, massd, momd, (real2*)posd);
        
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        
        CudaTest("kernel 3 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }

    /******************************************************************************/
    /*** sort bodies **************************************************************/
    /******************************************************************************/

    float cuSortKernel(
        int nnodesd, int nbodiesd,
        volatile int* __restrict sortd, const int* __restrict countd,
        volatile int* __restrict startd, volatile int* __restrict childd)
    {
        //fprintf(stderr, "SRKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);        
        hipEventRecord(start, 0);
                
        SortKernel << <blocks * FACTOR4, THREADS4 >> > (nnodesd, nbodiesd, sortd, countd, startd, childd);
        
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        
        CudaTest("kernel 4 launch failed");
 
        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** compute force ************************************************************/
    /******************************************************************************/

    float cuForceCalculationKernel(
        int nnodesd, int nbodiesd,
        int* __restrict errd,
        real itolsqd, real epssqd,
        const int* __restrict sortd, const int* __restrict childd,
        const moms* __restrict momd,
        const realPoint* __restrict posd,
        volatile realPoint* __restrict veld)
    {
        //fprintf(stderr, "FCKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        ForceCalculationKernel << <blocks * FACTOR5, THREADS5 >> > (nnodesd, nbodiesd, errd, itolsqd, epssqd, sortd, childd, momd, (real2*)posd, (real2*)veld);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        
        CudaTest("kernel 5 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** compute force (direct) ***************************************************/
    /******************************************************************************/

    float cuForceDirectCalculationKernel(
        int nnodesd, int nbodiesd,
        int* __restrict errd,
        real itolsqd, real epssqd,
        const int* __restrict sortd, const int* __restrict childd,
        const moms * __restrict momd,
        const realPoint * __restrict posd,
        volatile realPoint* __restrict veld)
    {
        //fprintf(stderr, "DFKernel\n");
        
        hipEvent_t startD, stopD;
        float timeD;

        hipEventCreate(&startD);  hipEventCreate(&stopD);
        hipEventRecord(startD, 0);
        
        ForceDirectCalculationKernel<<<(nbodiesd + BLOCKD - 1) / BLOCKD, BLOCKD>>> (nnodesd, nbodiesd, errd, itolsqd, epssqd, sortd, childd, momd, (real2*)posd, (real2*)veld);
        hipEventRecord(stopD, 0);  hipEventSynchronize(stopD);  hipEventElapsedTime(&timeD, startD, stopD);
        
        CudaTest("kernel direct launch failed");

        hipEventDestroy(startD);  hipEventDestroy(stopD);
        
        return timeD;
    }

}//namespace BHcu


