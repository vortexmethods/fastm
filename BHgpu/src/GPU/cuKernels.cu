#include "hip/hip_runtime.h"
/*--------------------------------*- BHgpu -*----------------*---------------*\
| #####   ##  ##                |                            | Version 1.5    |
| ##  ##  ##  ##   ####  ##  ## |  BHgpu: Barnes-Hut method  | 2023/08/29     |
| #####   ######  ##     ##  ## |  for 2D vortex particles   *----------------*
| ##  ##  ##  ##  ##     ##  ## |  Open Source Code                           |
| #####   ##  ##   ####   ####  |  https://www.github.com/vortexmethods/fastm |
|                                                                             |
| Copyright (C) 2020-2023 I. Marchevsky, E. Ryatina, A. Kolganova             |
| Copyright (C) 2013, Texas State University-San Marcos. All rights reserved. |
*-----------------------------------------------------------------------------*
| File name: cuKernels.cu                                                     |
| Info: Source code of BHgpu                                                  |
|                                                                             |
| This file is part of BHgpu.                                                 |
| BHcu is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| BHcu is distributed in the hope that it will be useful, but WITHOUT         |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with BHgpu.  If not, see <http://www.gnu.org/licenses/>.              |
\*---------------------------------------------------------------------------*/

/*!
\file
\brief Реализация CUDA-ядер
\author Марчевский Илья Константинович
\author Рятина Евгения Павловна
\author Колганова Александра Олеговна
\version 1.5
\date 29 августа 2023 г.
*/

#include "cuKernels.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include "operations.cuh"
#include "cuSort.cuh"

#include "Point2D.h"

#define WARPSIZE 32
#define MAXDEPTH 28
#define BLOCKD 32

#define codeLength 14
#define twoPowCodeLength (1 << codeLength)

namespace BHcu
{
    int blocks;
__device__ volatile int bottomd;
__device__ unsigned int blkcntd;
__device__ volatile real radiusd;

void setBlocks(int& blocks_)
{
     blocks_ = blocks;
}

void CudaSelect(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "There is no device supporting CUDA\n");
        exit(-1);
    }

    if ((dev < 0) || (deviceCount <= dev)) {
        fprintf(stderr, "There is no device %d\n", dev);
        exit(-1);
    }
    hipSetDevice(dev);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
        fprintf(stderr, "There is no CUDA capable device\n");
        exit(-1);
    }
    if (deviceProp.major < 2) {
        fprintf(stderr, "Need at least compute capability 2.0\n");
        exit(-1);
    }
    if (deviceProp.warpSize != WARPSIZE) {
        fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
        exit(-1);
    }

    blocks = deviceProp.multiProcessorCount;
    
    if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE - 1) != 0)) {
        fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
        exit(-1);
    }
    if (MAXDEPTH > WARPSIZE) {
        fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
        exit(-1);
    }
    if ((THREADS1 <= 0) || (THREADS1 & (THREADS1 - 1) != 0)) {
        fprintf(stderr, "THREADS1 must be greater than zero and a power of two\n");
        exit(-1);
    }

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);

    int fact = 1024;
    int driverVersion, runtimeVersion;

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("\n");
    printf("                          GPU Device Properties                         \n");
    printf("------------------------------------------------------------------------\n");
    printf("Name:                                  %s\n", properties.name); 
    printf("CUDA driver/runtime version:           %d.%d/%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("CUDA compute capabilitiy:              %d.%d\n", properties.major, properties.minor);
    printf("Number of multiprocessors:             %d\n", properties.multiProcessorCount);
    if (printFullCUDAinfo)
    {
        printf("GPU clock rate:                        %d (MHz)\n", properties.clockRate / fact);
        printf("Memory clock rate:                     %d (MHz)\n", properties.memoryClockRate / fact);
        printf("Memory bus width:                      %d-bit\n", properties.memoryBusWidth);
        printf("Theoretical memory bandwidth:          %d (GB/s)\n", (properties.memoryClockRate / fact * (properties.memoryBusWidth / 8) * 2) / fact);
        printf("Device global memory:                  %d (MB)\n", (int)(properties.totalGlobalMem / (fact * fact)));
        printf("Shared memory per block:               %d (KB)\n", (int)(properties.sharedMemPerBlock / fact));
        printf("Constant memory:                       %d (KB)\n", (int)(properties.totalConstMem / fact));
        printf("Maximum number of threads per block:   %d\n", properties.maxThreadsPerBlock);
        printf("Maximum thread dimension:              [%d, %d, %d]\n", properties.maxThreadsDim[0], properties.maxThreadsDim[1], properties.maxThreadsDim[2]);
        printf("Maximum grid size:                     [%d, %d, %d]\n", properties.maxGridSize[0], properties.maxGridSize[1], properties.maxGridSize[2]);
    }
    printf("------------------------------------------------------------------------\n");  
}


void cudaDelete(void* cudaPtr)
{
	hipFree(cudaPtr);
}


void* cudaNew(int n, size_t sizeType)
{
	void* cudaPtr;
	hipMalloc(&cudaPtr, sizeType * n);
	CudaTest("couldn't allocate device memory");

	return cudaPtr;
}

void cudaCopyVecToDevice(void* hostPtr, void* cudaPtr, size_t n, size_t typeSize)
{
	hipMemcpy(cudaPtr, hostPtr, typeSize * n, hipMemcpyHostToDevice);
	CudaTest("couldn't copy data from host to device");
}

void cudaCopyVecFromDevice(void* cudaPtr, void* hostPtr, size_t n, size_t typeSize)
{
	hipMemcpy(hostPtr, cudaPtr, typeSize * n, hipMemcpyDeviceToHost);
	CudaTest("couldn't copy data from device to host");
}


//////////////////
/// Error TEST
//////////////////


void CudaTest(const char* msg)
{
    hipError_t e;

    //hipDeviceSynchronize();
    hipDeviceSynchronize();
    if (hipSuccess != (e = hipGetLastError())) {
        fprintf(stderr, "%s: %d\n", msg, e);
        fprintf(stderr, "%s\n", hipGetErrorString(e));
        exit(-1);
    }
}


//////////////////
/// CUDA Kernels
//////////////////

__constant__ int binomCft[order * (order + 1)];

void setBinomCftConst(int* cft)
{
    hipMemcpyToSymbol(HIP_SYMBOL(binomCft), cft, order * (order + 1) * sizeof(int));
}



/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void InitializationKernel()
{
    blkcntd = 0;
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS1, FACTOR1)
void MBoundingBoxKernel(
    const int nbodiesd, 
    const real3* __restrict vtxd, 
    real2* __restrict Mposd, 
    volatile real2* __restrict maxrd, 
    volatile real2* __restrict minrd)
{
    register int i, j, k, inc;
    register real2 val;
    register real2 minr, maxr;
    __shared__ volatile real2 sminr[THREADS1], smaxr[THREADS1];

    // initialize with valid data (in case #bodies < #threads)
    minr.x = maxr.x = vtxd[0].x;
    minr.y = maxr.y = vtxd[0].y;

    // scan all bodies
    i = threadIdx.x;
    inc = THREADS1 * gridDim.x;
    for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
        val.x = vtxd[j].x;
        val.y = vtxd[j].y;

        minr.x = realmin(minr.x, val.x);
        maxr.x = realmax(maxr.x, val.x);

        minr.y = realmin(minr.y, val.y);
        maxr.y = realmax(maxr.y, val.y);
    }

    // reduction in shared memory
    sminr[i].x = minr.x;
    smaxr[i].x = maxr.x;
    sminr[i].y = minr.y;
    smaxr[i].y = maxr.y;

    for (j = THREADS1 / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sminr[i].x = minr.x = realmin(minr.x, sminr[k].x);
            smaxr[i].x = maxr.x = realmax(maxr.x, smaxr[k].x);
            sminr[i].y = minr.y = realmin(minr.y, sminr[k].y);
            smaxr[i].y = maxr.y = realmax(maxr.y, smaxr[k].y);
        }
    }

    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        minrd[k].x = minr.x;
        maxrd[k].x = maxr.x;
        minrd[k].y = minr.y;
        maxrd[k].y = maxr.y;

        

        __threadfence();

        inc = gridDim.x - 1;
        if (inc == atomicInc(&blkcntd, inc)) {
            // I'm the last block, so combine all block results
            for (j = 0; j <= inc; j++) {
                minr.x = realmin(minr.x, minrd[j].x);
                maxr.x = realmax(maxr.x, maxrd[j].x);
                minr.y = realmin(minr.y, minrd[j].y);
                maxr.y = realmax(maxr.y, maxrd[j].y);
            }

            // compute 'radius'
            radiusd = realmax(maxr.x - minr.x, maxr.y - minr.y) / 2;

            // create root node
            Mposd[0].x = (minr.x + maxr.x) / 2;
            Mposd[0].y = (minr.y + maxr.y) / 2;            
        }
    }
}

/******************************************************************************/
/*** Morton codes *************************************************************/
/******************************************************************************/

__global__
void MMortonCodesKernel (
    const int nbodies, 
    const real3* __restrict vtxd, 
    int* __restrict MmortonCodesKeyUnsortd, 
    int* __restrict MmortonCodesIdxUnsortd)
{
	int bdy = blockDim.x * blockIdx.x + threadIdx.x;

	if (bdy < nbodies)
	{
		real x = twoPowCodeLength * vtxd[bdy].x;
		real y = twoPowCodeLength * vtxd[bdy].y;

		unsigned int xx = MExpandBits((unsigned int)x);
		unsigned int yy = MExpandBits((unsigned int)y);
		MmortonCodesKeyUnsortd[bdy] = yy | (xx << 1);
		MmortonCodesIdxUnsortd[bdy] = bdy;
	}
}


/******************************************************************************/
/*** Morton Internal nodes tree build *****************************************/
/******************************************************************************/
__global__
void MMortonInternalNodesKernel(
    const int nbodies, 
    const int* __restrict MmortonCodesKeyd, 
    int* __restrict Mparentd, 
    int2* __restrict Mchildd, 
    int2* __restrict Mranged)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < nbodies - 1)
    {
        int d = sign(Delta(i, i + 1, nbodies, MmortonCodesKeyd) - Delta(i, i - 1, nbodies, MmortonCodesKeyd));
        int delta_min = Delta(i, i - d, nbodies, MmortonCodesKeyd);

        int Lmax = 2;
        while (Delta(i, i + Lmax * d, nbodies, MmortonCodesKeyd) > delta_min)
            Lmax *= 2;

        int L = 0;
        for (int t = (Lmax >> 1); t >= 1; t >>= 1)
            if (Delta(i, i + (L + t) * d, nbodies, MmortonCodesKeyd) > delta_min)
                L += t;

        int j = i + L * d;

        int delta_node = Delta(i, j, nbodies, MmortonCodesKeyd);

        int s = 0;
        for (int p = 1, t = ceilhalf(L); L > (1 << (p - 1)); ++p, t = ceilpow2(L, p))
        {
            int dl = Delta(i, i + (s + t) * d, nbodies, MmortonCodesKeyd);
            if (dl > delta_node)
                s += t;
        }//for p


        int gamma = i + s * d +   d * (d < 0);   //последнее слагаемое = std::min(d, 0);

        int Mmin = min(i, j);
        int Mmax = max(i, j);
        
        const int& left = gamma;
        const int& right = gamma + 1;

        // Левый потомок - лист или внутренний узел
        int childLeft = Mchildd[i].x = (Mmin == gamma) * nbodies + left;
        
        Mranged[childLeft].x = Mmin;
        Mranged[childLeft].y = gamma;
        Mparentd[childLeft] = i;

        // Правый потомок - лист или внутренний узел
        int childRight = Mchildd[i].y = (Mmax == gamma + 1) * nbodies + right;

        Mranged[childRight].x = gamma+1;
        Mranged[childRight].y = Mmax;
        Mparentd[childRight] = i;
    }
}

/******************************************************************************/
/*** Morton Internal nodes geometry calculation *******************************/
/******************************************************************************/
__global__
void MMortonInternalCellsGeometryKernel(
    const int nbodies,
    const int* __restrict MmortonCodesKeyd,
    real2* __restrict Mposd,
    real2* __restrict Msized,
    const int2* __restrict Mranged,
    int* __restrict MlevelUnsortd,
    int* __restrict MindexUnsortd
)
{
    int cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (cell < nbodies - 1)
    {
        int prLength = min(Delta(Mranged[cell].x, Mranged[cell].y, nbodies, MmortonCodesKeyd), 2 * codeLength);
        unsigned int pr = (MmortonCodesKeyd[Mranged[cell].x] >> (2 * codeLength - prLength));
               
        prLength -= min(Delta(Mranged[0].x, Mranged[0].y, nbodies, MmortonCodesKeyd), 2 * codeLength);
        
        real2 sz;
        sz.x = 1 / (real)(1 << ceilhalf(prLength));
        sz.y = 1 / (real)(1 << (prLength / 2));

        real2 pos;
        pos.x = sz.x / 2;
        pos.y = sz.y / 2;

        int xint = MShrinkBits(pr);       
        int yint = MShrinkBits(pr >> 1);

        real addX = xint * sz.x;
        real addY = yint * sz.y;

        if (prLength & 1)
        {
            pos.x += addX;
            pos.y += addY;
        }
        else
        {
            pos.y += addX;
            pos.x += addY;
        }			          

        Mposd[cell] = pos;   
        Msized[cell] = sz;    

                
        MlevelUnsortd[cell] = prLength;
        MindexUnsortd[cell] = cell;
    }

}//MMortonInternalCellsGeometryKernel(...)


/******************************************************************************/
/*** permutation list transposition *******************************************/
/******************************************************************************/
__global__
void MTransposeIndexKernel(
    const int nbodiesd, const int nnodesd,
    const int* __restrict MindexSortd, 
    int* __restrict MindexSortTd)
{
    register const int cell = blockDim.x * blockIdx.x + threadIdx.x;
    register const int newcell = MindexSortd[cell];

    if (cell < nbodiesd - 1)
        MindexSortTd[newcell] = cell;       

}//MTransposeIndexKernel



/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(1024, 1)
void ClearKernel2(
    const int nnodesd, const int nbodiesd, 
    volatile int* __restrict massd)
{
    register int k, inc, bottom;

    bottom = nnodesd - (nbodiesd - 1); //bottomd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    if (k < bottom) k += inc;

// 0 1 ... (nb-1)  (nb+0) ... (nb+nb-2)
// --------------  --------------------
//     bodies              cells

    // iterate over all cells assigned to thread
    while (k < nnodesd) 
	{
        massd[nnodesd - 1 - k] = -1;
        k += inc;
    }
}



/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/
#include "ShiftKernels/IncludeKer.cu"

/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS5, FACTOR5)
void ForceCalculationKernel2(
    const int nnodesd, const int nbodiesd,
    const real itolsqd, const real epssqd,
    const int2* __restrict Mchildd,
    const real2* __restrict momsd,
    const real3* __restrict vtxd,    
    const int* __restrict MmortonCodesIdxd,
    const real2* __restrict Mposd, const int* __restrict MindexSortd, const int* __restrict MindexSortTd,
    real2* __restrict veld,  //veld - без volatile
    const real2* __restrict Msized)

{
    register int j, k, n, depth, base, sbase, pd, nd;
    register real2 p, v, dr, ps;
    register real r2;
    register const real2* mom;
    //register real2 mom[order];  
    //register real2 mom0, mom1, mom2, mom3, mom4, mom5, mom6, mom7, mom8, mom9, mom10, mom11, mom12, mom13;

    register real2 th;
    

    __shared__ volatile int pos[MAXDEPTH * THREADS5 / WARPSIZE], node[MAXDEPTH * THREADS5 / WARPSIZE];



    // figure out first thread in each warp (lane 0)
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;
    //diff = threadIdx.x - sbase;

    __syncthreads();
    __threadfence_block();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x)
    {
        const int indexInParticles = MmortonCodesIdxd[k];
        p = real2{ vtxd[indexInParticles].x, vtxd[indexInParticles].y };

        v.x = 0;
        v.y = 0;

        // initialize iteration stack, i.e., push root node onto stack
        depth = j;
        if (sbase == threadIdx.x)
        {
            pos[j] = 0;
            node[j] = nnodesd - 1;
        }

        do
        {
            // stack is not empty
            pd = pos[depth];
            nd = node[depth];

            register int2 chBoth = Mchildd[MindexSortd[(nnodesd - 1) - nd]];

			register real gm;
			register real2 sumSide2;
			bool isVortex;

            while (pd < 2)
            {
                // node on top of stack has more children to process

                // load child pointer
                //computation of n = childd[nd + pd] (pd = 0 или pd = 1)
				int chd = pd * chBoth.y + (1-pd) * chBoth.x;
				++pd;
				
				isVortex = (chd >= nbodiesd);
				
				if (isVortex)
				{
					n = chd - nbodiesd;
					ps = real2{ vtxd[MmortonCodesIdxd[n]].x, vtxd[MmortonCodesIdxd[n]].y };
					gm = vtxd[MmortonCodesIdxd[n]].z;
					sumSide2 = real2{ (real)0, (real)0 };
				}
				else
				{
					register const int srtT = MindexSortTd[chd];
					n = (nnodesd - 1) - srtT;
					ps = Mposd[chd];
					mom = momsd + (srtT * order);
					gm = mom[0].x;
					sumSide2 = Msized[chd];
				}

				dr = p - ps;

				//for (i = 0; i < order; ++i)
				//    mom[i] = momsd[n * order + i];

				//mom0 = momsd[n * order];
				//mom1 = momsd[n * order + 1];
				//mom2 = momsd[n * order + 2];
				//mom3 = momsd[n * order + 3];
				//mom4 = momsd[n * order + 4];
				//mom5 = momsd[n * order + 5];
				//mom6 = momsd[n * order + 6];
				//mom7 = momsd[n * order + 7];
				//mom8 = momsd[n * order + 8];
				//mom9 = momsd[n * order + 9];
				//mom10 = momsd[n * order + 10];
				//mom11 = momsd[n * order + 11];
				//mom12 = momsd[n * order + 12];
				//mom13 = momsd[n * order + 13];

				r2 = (dr.x * dr.x + dr.y * dr.y);   // compute distance squared

				// check if all threads agree that cell is far enough away (or is a body)
				if (isVortex || __all_sync(0xffffffff, ((sumSide2.x+sumSide2.y)*(sumSide2.x+sumSide2.y) + epssqd) * itolsqd < r2))
				{  
#ifdef CALCinDOUBLE
					real f = gm / realmax(r2, epssqd);
#else
					real f = fdividef(gm, realmax(r2,epssqd));
#endif
					v += f * dr;

					if ((!isVortex) && (order > 1))
					{
#ifdef CALCinDOUBLE
						real2 cftr = (r2 ? (1.0 / r2) : (real)0) * dr;
#else
						real2 cftr = (r2 ? fdividef(1.0f, r2) : 0.0f) * dr;
#endif
						th = cftr;
						
						for (int s = 1; s < order; ++s)
						{
							th = multz(th, cftr);
#ifdef CALCinFLOAT                                    
							if (isinf(th.x) || isinf(th.y))
							{
								//printf("s = %d\n", s);
								break;
							}
#endif
							v += multzA(th, mom[s]);
						}
						
						//th = multz(th, cftr);
						//v += multzA(th, mom1);

						//th = multz(th, cftr);
						//v += multzA(th, mom2);

						//th = multz(th, cftr);
						//v += multzA(th, mom3);

						//th = multz(th, cftr);
						//v += multzA(th, mom4);

						//th = multz(th, cftr);
						//v += multzA(th, mom5);

						//th = multz(th, cftr);
						//v += multzA(th, mom6);

						//th = multz(th, cftr);
						//v += multzA(th, mom7);

						//th = multz(th, cftr);
						//v += multzA(th, mom8);

						//th = multz(th, cftr);
						//v += multzA(th, mom9);

						//th = multz(th, cftr);
						//v += multzA(th, mom10);

						//th = multz(th, cftr);
						//v += multzA(th, mom11);

						//th = multz(th, cftr);
						//v += multzA(th, mom12);

						//th = multz(th, cftr);
						//v += multzA(th, mom13);
					}
				}
				else
				{
					// push cell onto stack
					if (sbase == threadIdx.x)
					{  // maybe don't push and inc if last child
						pos[depth] = pd;
						node[depth] = nd;
					}
					depth++;
					pd = 0;
					nd = n;

					chBoth = Mchildd[MindexSortd[(nnodesd - 1) - nd]];
				}
                
            }
            depth--;  // done with this level
        } while (depth >= j);

        // update velocity

        real2 result = real2{ -v.y, v.x };
        veld[indexInParticles] = result;        
    }
} 


/******************************************************************************/
/*** compute force (direct) ***************************************************/
/******************************************************************************/


__global__
//__launch_bounds__(THREADSD, FACTORD)
void ForceDirectCalculationKernel(
    const int nnodesd, const int nbodiesd,
    const real epssqd,
    const real3* __restrict vtxd,    
    real2* __restrict veld)
{
    __shared__ real3 shvtx[BLOCKD];    

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    real2 pt;
    pt.x = vtxd[i].x;
    pt.y = vtxd[i].y;

    real2 vel;
    vel.x = vel.y = 0;

    real2 dr;
    real dr2, izn;

    //vortices
    for (size_t j = 0; j < nbodiesd; j += BLOCKD)
    {
        shvtx[threadIdx.x] = vtxd[j + threadIdx.x];               

        __syncthreads();

        for (size_t q = 0; q < BLOCKD; ++q)
        {
            if (j + q < nbodiesd)
            {
                dr.x = pt.x - shvtx[q].x;
                dr.y = pt.y - shvtx[q].y;
                dr2 = dr.x * dr.x + dr.y * dr.y;

                izn = shvtx[q].z / realmax(dr2, epssqd);// / CUboundDenom(dr2, eps2); //РЎРіР»Р°Р¶РёРІР°С‚СЊ РЅР°РґРѕ!!!

                vel.x -= dr.y * izn;
                vel.y += dr.x * izn;

            }
        }
        __syncthreads();
    }

    if (i < nbodiesd)
    {
        veld[i].x = vel.x;// * iDPI;
        veld[i].y = vel.y;// * iDPI;
    }
    //*/
}


/******************************************************************************/


void KernelsOptimization()
{
    // set L1/shared memory configuration

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value
        
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceDirectCalculationKernel), hipFuncCachePreferEqual); //d
    hipGetLastError();  // reset error value
        
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel2_14), hipFuncCachePreferL1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(SummarizationKernel2_16), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceCalculationKernel2), hipFuncCachePreferL1); //d
    hipGetLastError();  // reset error value

}


/******************************************************************************/





//////////////////
/// Wrappers
//////////////////



    /******************************************************************************/
    /*** initialize memory ********************************************************/
    /******************************************************************************/

    float cuInitializationKernel()
    {
        //fprintf(stderr, "IKKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);

        hipEventRecord(start, 0);
        InitializationKernel<<<1, 1>>> ();
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        CudaTest("kernel 0 launch failed");
        
        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }


    /******************************************************************************/
    /*** compute center and radius ************************************************/
    /******************************************************************************/
	float McuBoundingBoxKernel(
		int nbodiesd,
		const realVortex* __restrict vtxd,
		realPoint* __restrict Mposd,
		volatile realPoint* __restrict maxrd, volatile realPoint* __restrict minrd)
	{
		hipEvent_t start, stop;
		float time;

		hipEventCreate(&start);  hipEventCreate(&stop);
		hipEventRecord(start, 0);

		MBoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>> (nbodiesd, (real3*)vtxd, (real2*)Mposd, (real2*)maxrd, (real2*)minrd);
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

		CudaTest("Mkernel 1 launch failed");

		hipEventDestroy(start);  hipEventDestroy(stop);
		return time;
	}

	/******************************************************************************/
	/*** Morton codes *************************************************************/
	/******************************************************************************/

    float McuMortonCodesKernel(
        int nbodiesd,
        realVortex* __restrict vtxd,
        int* __restrict MmortonCodesKeyUnsortd, int* __restrict MmortonCodesIdxUnsortd,
        int* __restrict MmortonCodesKeyd, int* __restrict MmortonCodesIdxd,
        intPair* __restrict Mranged)
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = (nbodiesd + 31) / 32;
        dim3 Mthreads = 32;

        MMortonCodesKernel << <Mblocks, Mthreads >> > (nbodiesd, (real3*)vtxd, MmortonCodesKeyUnsortd, MmortonCodesIdxUnsortd);


        ///RadixSort

        RadixSortFromCUB(
            MmortonCodesKeyUnsortd, MmortonCodesKeyd, \
            MmortonCodesIdxUnsortd, MmortonCodesIdxd, \
            nbodiesd, 0, 2 * codeLength);


        //Заполнение нулевой ячейки (диапазон для корня дерева)
        int totalRange[2] = { 0, nbodiesd - 1 };
        cudaCopyVecToDevice(totalRange, Mranged, 2, sizeof(int));

		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

		CudaTest("Mkernel 1 launch failed");

		hipEventDestroy(start);  hipEventDestroy(stop);

        

		return time;
	}

    /******************************************************************************/
    /*** Morton Internal nodes build **********************************************/
    /******************************************************************************/

    float McuMortonInternalNodesKernel(
        int nbodiesd,
        int* __restrict MmortonCodesKeyd, 
        int* __restrict Mparentd,
        intPair* __restrict Mchildd,
        intPair* __restrict Mranged
    )
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = ((nbodiesd - 1) + 31) / 32;
        dim3 Mthreads = 32;

        MMortonInternalNodesKernel<<<Mblocks, Mthreads>>> (nbodiesd, MmortonCodesKeyd, Mparentd, (int2*)Mchildd, (int2*)Mranged);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("Mkernel 2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;

    }



    /******************************************************************************/
    /*** Morton Internal nodes geometry calculation *******************************/
    /******************************************************************************/
    float McuMortonInternalCellsGeometryKernel(
        int nbodiesd,
        int nnodesd,
        int* __restrict MmortonCodesKeyd,
        realPoint* __restrict Mposd,
        realPoint* __restrict Msized,
        intPair* __restrict Mranged,
        int* __restrict MlevelUnsortd,
        int* __restrict MlevelSortd,
        int* __restrict MindexUnsortd,
        int* __restrict MindexSortd,
        int* __restrict MindexSortTd
    )
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = ((nbodiesd) + 31) / 32;
        dim3 Mthreads = 32;

		MMortonInternalCellsGeometryKernel<<<Mblocks, Mthreads>>>(nbodiesd, MmortonCodesKeyd, (real2*)Mposd, (real2*)Msized, (int2*)Mranged, MlevelUnsortd, MindexUnsortd);

        RadixSortFromCUB( \
            MlevelUnsortd, MlevelSortd, \
            MindexUnsortd, MindexSortd, \
            nbodiesd-1, 0, 2 * codeLength);

        MTransposeIndexKernel << <Mblocks, Mthreads >> > (nbodiesd, nnodesd, MindexSortd, MindexSortTd);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("Mkernel 3 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }



	 	 

    /******************************************************************************/
    /*** build tree ***************************************************************/
    /******************************************************************************/   

    float cuClearKernel2(
        int nnodesd, int nbodiesd,
        volatile int* __restrict massd,        
        volatile realPoint* __restrict momsd)
    {
        //fprintf(stderr, "CxKernel\n");
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        hipMemset((void*)momsd, 0, (nbodiesd-1) * order * sizeof(realPoint));

        ClearKernel2 << <blocks * 1, 1024 >> > (nnodesd, nbodiesd, massd);
        
	       	

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel clear2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** compute multipole moments for all the cells ******************************/
    /******************************************************************************/
    float cuSummarizationKernel2(
        const int nnodesd, const int nbodiesd,
        const intPair* __restrict Mchildd,
        volatile int* __restrict massd,
        volatile realPoint* __restrict momsd,
        const realVortex* __restrict vtxd, const int* __restrict MmortonCodesIdxd,
        const realPoint* __restrict Mposd, const int* __restrict MindexSortd, const int* __restrict MindexSortTd)
    {
        //fprintf(stderr, "SKKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
#include "ShiftKernels/SwitchKer.cu"

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 3 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }

   

    /******************************************************************************/
    /*** compute force ************************************************************/
    /******************************************************************************/
    float cuForceCalculationKernel2(
        int nnodesd, int nbodiesd,
        real itolsqd, real epssqd,
        const intPair* __restrict Mchildd,
        const realPoint* __restrict momsd,
        const realVortex* __restrict vtxd, const int* __restrict MmortonCodesIdxd,
        const realPoint* __restrict Mposd, const int* __restrict MindexSortd, const int* __restrict MindexSortTd,
        volatile realPoint* __restrict veld,
        const realPoint* __restrict Msized)
    {
        //fprintf(stderr, "FCKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        ForceCalculationKernel2 << <blocks * FACTOR5, THREADS5 >> > (
            nnodesd, nbodiesd, itolsqd, epssqd, (int2*)Mchildd, (real2*)momsd,
            (real3*)vtxd, MmortonCodesIdxd,
            (real2*)Mposd, MindexSortd, MindexSortTd,
            (real2*)veld, (real2*)Msized);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 5 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }

    /******************************************************************************/
    /*** compute force (direct) ***************************************************/
    /******************************************************************************/

    float cuForceDirectCalculationKernel(
        int nnodesd, int nbodiesd,
        real epssqd,
        const realVortex* __restrict vtxd,
        volatile realPoint* __restrict veld)
    {
        //fprintf(stderr, "DFKernel\n");
        
        hipEvent_t startD, stopD;
        float timeD;

        hipEventCreate(&startD);  hipEventCreate(&stopD);
        hipEventRecord(startD, 0);
        
        ForceDirectCalculationKernel<<<(nbodiesd + BLOCKD - 1) / BLOCKD, BLOCKD>>> (nnodesd, nbodiesd, epssqd, (real3*)vtxd, (real2*)veld);
        hipEventRecord(stopD, 0);  hipEventSynchronize(stopD);  hipEventElapsedTime(&timeD, startD, stopD);
        
        CudaTest("kernel direct launch failed");

        hipEventDestroy(startD);  hipEventDestroy(stopD);
        
        return timeD;
    }

}//namespace BHcu