#include "hip/hip_runtime.h"
/*--------------------------------*- BHgpu -*----------------*---------------*\
| #####   ##  ##                |                            | Version 1.4    |
| ##  ##  ##  ##   ####  ##  ## |  BHgpu: Barnes-Hut method  | 2023/03/28     |
| #####   ######  ##     ##  ## |  for 2D vortex particles   *----------------*
| ##  ##  ##  ##  ##     ##  ## |  Open Source Code                           |
| #####   ##  ##   ####   ####  |  https://www.github.com/vortexmethods/fastm |
|                                                                             |
| Copyright (C) 2020-2023 I. Marchevsky, E. Ryatina, A. Kolganova             |
| Copyright (C) 2013, Texas State University-San Marcos. All rights reserved. |
*-----------------------------------------------------------------------------*
| File name: cuKernels.cu                                                     |
| Info: Source code of BHgpu                                                  |
|                                                                             |
| This file is part of BHgpu.                                                 |
| BHcu is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| BHcu is distributed in the hope that it will be useful, but WITHOUT         |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with BHgpu.  If not, see <http://www.gnu.org/licenses/>.              |
\*---------------------------------------------------------------------------*/

/*!
\file
\brief Реализация CUDA-ядер
\author Марчевский Илья Константинович
\author Рятина Евгения Павловна
\author Колганова Александра Олеговна
\version 1.4
\date 28 марта 2023 г.
*/

#include "cuKernels.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include "operations.cuh"
#include "cuSort.cuh"

#include "Point2D.h"

#define WARPSIZE 32
#define MAXDEPTH 32
#define BLOCKD 32

#define codeLength 14
#define twoPowCodeLength (1 << codeLength)

namespace BHcu
{
    int blocks;
__device__ volatile int bottomd, maxdepthd;
__device__ unsigned int blkcntd;
__device__ volatile real radiusd;

void setBlocks(int& blocks_)
{
     blocks_ = blocks;
}

void CudaSelect(int dev)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "There is no device supporting CUDA\n");
        exit(-1);
    }

    if ((dev < 0) || (deviceCount <= dev)) {
        fprintf(stderr, "There is no device %d\n", dev);
        exit(-1);
    }
    hipSetDevice(dev);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
        fprintf(stderr, "There is no CUDA capable device\n");
        exit(-1);
    }
    if (deviceProp.major < 2) {
        fprintf(stderr, "Need at least compute capability 2.0\n");
        exit(-1);
    }
    if (deviceProp.warpSize != WARPSIZE) {
        fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
        exit(-1);
    }

    blocks = deviceProp.multiProcessorCount;
    
    if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE - 1) != 0)) {
        fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
        exit(-1);
    }
    if (MAXDEPTH > WARPSIZE) {
        fprintf(stderr, "MAXDEPTH must be less than or equal to WARPSIZE\n");
        exit(-1);
    }
    if ((THREADS1 <= 0) || (THREADS1 & (THREADS1 - 1) != 0)) {
        fprintf(stderr, "THREADS1 must be greater than zero and a power of two\n");
        exit(-1);
    }

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);

    int fact = 1024;
    int driverVersion, runtimeVersion;

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("\n");
    printf("                          GPU Device Properties                         \n");
    printf("------------------------------------------------------------------------\n");
    printf("Name:                                  %s\n", properties.name); 
    printf("CUDA driver/runtime version:           %d.%d/%d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("CUDA compute capabilitiy:              %d.%d\n", properties.major, properties.minor);
    printf("Number of multiprocessors:             %d\n", properties.multiProcessorCount);
    if (printFullCUDAinfo)
    {
        printf("GPU clock rate:                        %d (MHz)\n", properties.clockRate / fact);
        printf("Memory clock rate:                     %d (MHz)\n", properties.memoryClockRate / fact);
        printf("Memory bus width:                      %d-bit\n", properties.memoryBusWidth);
        printf("Theoretical memory bandwidth:          %d (GB/s)\n", (properties.memoryClockRate / fact * (properties.memoryBusWidth / 8) * 2) / fact);
        printf("Device global memory:                  %d (MB)\n", (int)(properties.totalGlobalMem / (fact * fact)));
        printf("Shared memory per block:               %d (KB)\n", (int)(properties.sharedMemPerBlock / fact));
        printf("Constant memory:                       %d (KB)\n", (int)(properties.totalConstMem / fact));
        printf("Maximum number of threads per block:   %d\n", properties.maxThreadsPerBlock);
        printf("Maximum thread dimension:              [%d, %d, %d]\n", properties.maxThreadsDim[0], properties.maxThreadsDim[1], properties.maxThreadsDim[2]);
        printf("Maximum grid size:                     [%d, %d, %d]\n", properties.maxGridSize[0], properties.maxGridSize[1], properties.maxGridSize[2]);
    }
    printf("------------------------------------------------------------------------\n");  
}


void cudaDelete(void* cudaPtr)
{
	hipFree(cudaPtr);
}


void* cudaNew(int n, size_t sizeType)
{
	void* cudaPtr;
	hipMalloc(&cudaPtr, sizeType * n);
	CudaTest("couldn't allocate device memory");

	return cudaPtr;
}

void cudaCopyVecToDevice(void* hostPtr, void* cudaPtr, size_t n, size_t typeSize)
{
	hipMemcpy(cudaPtr, hostPtr, typeSize * n, hipMemcpyHostToDevice);
	CudaTest("couldn't copy data from host to device");
}

void cudaCopyVecFromDevice(void* cudaPtr, void* hostPtr, size_t n, size_t typeSize)
{
	hipMemcpy(hostPtr, cudaPtr, typeSize * n, hipMemcpyDeviceToHost);
	CudaTest("couldn't copy data from device to host");
}


//////////////////
/// Error TEST
//////////////////


void CudaTest(const char* msg)
{
    hipError_t e;

    //hipDeviceSynchronize();
    hipDeviceSynchronize();
    if (hipSuccess != (e = hipGetLastError())) {
        fprintf(stderr, "%s: %d\n", msg, e);
        fprintf(stderr, "%s\n", hipGetErrorString(e));
        exit(-1);
    }
}


//////////////////
/// CUDA Kernels
//////////////////

/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void InitializationKernel(int* __restrict errd)
{
    *errd = 0;
    maxdepthd = 1;
    blkcntd = 0;
}


/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS1, FACTOR1)
void MBoundingBoxKernel(int nbodiesd, volatile real2* __restrict posd, volatile real2* __restrict Mposd, volatile real2* __restrict maxrd, volatile real2* __restrict minrd)
{
    register int i, j, k, inc;
    register real2 val;
    register real2 minr, maxr;
    __shared__ volatile real2 sminr[THREADS1], smaxr[THREADS1];

    // initialize with valid data (in case #bodies < #threads)
    minr.x = maxr.x = posd[0].x;
    minr.y = maxr.y = posd[0].y;

    // scan all bodies
    i = threadIdx.x;
    inc = THREADS1 * gridDim.x;
    for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
        val.x = posd[j].x;
        val.y = posd[j].y;

        minr.x = realmin(minr.x, val.x);
        maxr.x = realmax(maxr.x, val.x);

        minr.y = realmin(minr.y, val.y);
        maxr.y = realmax(maxr.y, val.y);
    }

    // reduction in shared memory
    sminr[i].x = minr.x;
    smaxr[i].x = maxr.x;
    sminr[i].y = minr.y;
    smaxr[i].y = maxr.y;

    for (j = THREADS1 / 2; j > 0; j /= 2) {
        __syncthreads();
        if (i < j) {
            k = i + j;
            sminr[i].x = minr.x = realmin(minr.x, sminr[k].x);
            smaxr[i].x = maxr.x = realmax(maxr.x, smaxr[k].x);
            sminr[i].y = minr.y = realmin(minr.y, sminr[k].y);
            smaxr[i].y = maxr.y = realmax(maxr.y, smaxr[k].y);
        }
    }

    // write block result to global memory
    if (i == 0) {
        k = blockIdx.x;
        minrd[k].x = minr.x;
        maxrd[k].x = maxr.x;
        minrd[k].y = minr.y;
        maxrd[k].y = maxr.y;

        

        __threadfence();

        inc = gridDim.x - 1;
        if (inc == atomicInc(&blkcntd, inc)) {
            // I'm the last block, so combine all block results
            for (j = 0; j <= inc; j++) {
                minr.x = realmin(minr.x, minrd[j].x);
                maxr.x = realmax(maxr.x, maxrd[j].x);
                minr.y = realmin(minr.y, minrd[j].y);
                maxr.y = realmax(maxr.y, maxrd[j].y);
            }

            // compute 'radius'
            radiusd = realmax(maxr.x - minr.x, maxr.y - minr.y) / 2;

            // create root node
            k = 0;

            Mposd[k].x = (minr.x + maxr.x) / 2;
            Mposd[k].y = (minr.y + maxr.y) / 2;            
        }
    }
}

/******************************************************************************/
/*** Morton codes *************************************************************/
/******************************************************************************/

__global__
void MMortonCodesKernel (int nbodies, volatile real2* __restrict posd, 
    volatile int* __restrict MmortonCodesKeyUnsortd, volatile int* __restrict MmortonCodesIdxUnsortd)
{
	int bdy = blockDim.x * blockIdx.x + threadIdx.x;

	if (bdy < nbodies)
	{
		real x = twoPowCodeLength * posd[bdy].x;
		real y = twoPowCodeLength * posd[bdy].y;

		unsigned int xx = MExpandBits((unsigned int)x);
		unsigned int yy = MExpandBits((unsigned int)y);
		MmortonCodesKeyUnsortd[bdy] = yy | (xx << 1);
		MmortonCodesIdxUnsortd[bdy] = bdy;
	}
}

/******************************************************************************/
/*** Morton Internal nodes tree build *****************************************/
/******************************************************************************/
__global__
void MMortonInternalNodesKernel(
    int nbodies, 
    int* __restrict MmortonCodesKeyd, 
    int* __restrict Mparentd, 
    int2* __restrict Mchildd, 
    int2* __restrict Mranged)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < nbodies - 1)
    {
        int d = sign(Delta(i, i + 1, nbodies, MmortonCodesKeyd) - Delta(i, i - 1, nbodies, MmortonCodesKeyd));
        int delta_min = Delta(i, i - d, nbodies, MmortonCodesKeyd);

        int Lmax = 2;
        while (Delta(i, i + Lmax * d, nbodies, MmortonCodesKeyd) > delta_min)
            Lmax *= 2;

        int L = 0;
        for (int t = (Lmax >> 1); t >= 1; t >>= 1)
            if (Delta(i, i + (L + t) * d, nbodies, MmortonCodesKeyd) > delta_min)
                L += t;

        int j = i + L * d;

        int delta_node = Delta(i, j, nbodies, MmortonCodesKeyd);

        int s = 0;
        for (int p = 1, t = ceilhalf(L); L > (1 << (p - 1)); ++p, t = ceilpow2(L, p))
        {
            int dl = Delta(i, i + (s + t) * d, nbodies, MmortonCodesKeyd);
            if (dl > delta_node)
                s += t;
        }//for p


        int gamma = i + s * d +    d * (d < 0);   //последнее слагаемое = std::min(d, 0);

        int Mmin = min(i, j);
        int Mmax = max(i, j);
        
        const int& left = gamma;
        const int& right = gamma + 1;

        // Левый потомок - лист или внутренний узел
        int childLeft = Mchildd[i].x = (Mmin == gamma) * nbodies + left;
        
        Mranged[childLeft].x = Mmin;
        Mranged[childLeft].y = gamma;
        Mparentd[childLeft] = i;

        // Правый потомок - лист или внутренний узел
        int childRight = Mchildd[i].y = (Mmax == gamma + 1) * nbodies + right;

        Mranged[childRight].x = gamma+1;
        Mranged[childRight].y = Mmax;
        Mparentd[childRight] = i;
    }
}

/******************************************************************************/
/*** Morton Internal nodes geometry calculation *******************************/
/******************************************************************************/
__global__
void MMortonInternalCellsGeometryKernel(
    int nbodies,
    int* __restrict MmortonCodesKeyd,
    real2* __restrict Mposd,
    real2* __restrict Msized,
    int2* __restrict Mranged,
    int* __restrict MlevelUnsortd,
    int* __restrict MindexUnsortd
)
{
    int cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (cell < nbodies - 1)
    {
        int prLength = min(Delta(Mranged[cell].x, Mranged[cell].y, nbodies, MmortonCodesKeyd), 2 * codeLength);
        unsigned int pr = (MmortonCodesKeyd[Mranged[cell].x] >> (2 * codeLength - prLength));
               
        prLength -= min(Delta(Mranged[0].x, Mranged[0].y, nbodies, MmortonCodesKeyd), 2 * codeLength);
        
        real2 sz;
        sz.x = 1 / (real)(1 << ceilhalf(prLength));
        sz.y = 1 / (real)(1 << (prLength / 2));

        real2 pos;
        pos.x = sz.x / 2;
        pos.y = sz.y / 2;

        int xint = MShrinkBits(pr);       
        int yint = MShrinkBits(pr >> 1);

        real addX = xint * sz.x;
        real addY = yint * sz.y;

        if (prLength & 1)
        {
            pos.x += addX;
            pos.y += addY;
        }
        else
        {
            pos.y += addX;
            pos.x += addY;
        }			          

        Mposd[cell] = pos;   
        Msized[cell] = sz;    

                
        MlevelUnsortd[cell] = prLength;
        MindexUnsortd[cell] = cell;
    }

}//MMortonInternalCellsGeometryKernel(...)




/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(1024, 1)
void ClearKernel2(int nnodesd, int nbodiesd, volatile int* __restrict startd, volatile int* __restrict massd)
{
    register int k, inc, bottom;

    bottom = nnodesd - (nbodiesd - 2); //bottomd;
    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    if (k < bottom) k += inc;

    // iterate over all cells assigned to thread
    while (k < nnodesd) {
        massd[k] = -1;
        startd[k] = -1;
        k += inc;
    }
}


__global__
__launch_bounds__(1024, 1)
void ClearKernel3(int nnodesd, int nbodies, const real* __restrict gamd, volatile real* __restrict momsd)
{
    register int k, inc;

    inc = blockDim.x * gridDim.x;
    k = threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    
    // iterate over all cells assigned to thread
    while (k < nnodesd) {   
          momsd[k * (order * 2 - 1)] = (k < nbodies) ? gamd[k] : 0;
          for(int s = 1; s < (order * 2 - 1); ++s)
            momsd[k * (order * 2 - 1) + s] = 0;

        k += inc;
    }
}

/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS3, FACTOR3)
void SummarizationKernel2(const int nnodesd, const int nbodiesd, volatile int* __restrict countd, const int* __restrict childd, volatile int* __restrict massd, volatile real* __restrict momsd, volatile real2* __restrict posd,
    const int* __restrict cftl)
{
    register int i, j, k, ch, inc, cnt, bottom, flag;

    register real mom[order * 2 - 1];
    //register volatile real* mom;
    
    register real cmom[order * 2 - 1];
    
    register real2 momh[order-1]; // (order-1) !!!
    register real2 cen, dr;

    register int m, cm;

    register real binCft;

    bottom = nnodesd - (nbodiesd - 2);// bottomd;
      
    __syncthreads();

    inc = blockDim.x * gridDim.x;
    k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
    
    if (k < bottom) k += inc;

    //register int restart = k;

    flag = 0;
    j = 0;
    // iterate over all cells assigned to thread
    while (k <= nnodesd) {
        if (massd[k] >= 0) {
            k += inc;
        }
        else {
            if (j == 0) {
                j = 2;
                for (i = 0; i < 2; i++) {
                    ch = childd[k * 2 + i];
                    if ((ch < nbodiesd) || (massd[ch] >= 0)) {
                        j--;
                    }

                }
            }
            else {
                j = 2;
                for (i = 0; i < 2; i++) {
                    ch = childd[k * 2 + i];                    

                    if ((ch < nbodiesd) || (massd[ch] >= 0))
                    {
                        j--;
                    }
                }
            }

            if (j == 0) {
                // all children are ready

                cm = 0;

                for (int s = 0; s < (order * 2 - 1); ++s)
                    cmom[s] = 0;

                if (order > 1)
                {
                    cen.x = posd[k].x;
                    cen.y = posd[k].y;
                }


                cnt = 0;
                for (i = 0; i < 2; i++) {
                    ch = childd[k * 2 + i];                   

                    if (ch >= 0) {
                        for (int s = 0; s < (order * 2 - 1); ++s)
                            mom[s] = momsd[ch * (order * 2 - 1) + s];
                        //mom = momsd + (ch * (order * 2 - 1));

                        if (order > 1)
                        {
                            dr.x = posd[ch].x - cen.x; 
                            dr.y = posd[ch].y - cen.y;
                        }
                        
                        m = massd[ch];

                        cnt += (ch >= nbodiesd) ? countd[ch] : 1;

                        // add child's contribution    

                        cmom[0] += mom[0];

                        for (int p = 1; p < 2 * (order - 1); p += 2)
                        {
                            cmom[p + 0] += mom[p + 0];
                            cmom[p + 1] += mom[p + 1];

                            for (int q = 0; q < (p - 1) / 2; ++q)
                                momh[q] = multz(momh[q], dr);                                

                            if (p==1)
                              momh[0] = mom[0] * dr;
                            else
                              momh[(int)(p - 1) / 2] = multz(mom[p - 2], mom[p - 1], dr);
                            
                            cmom[p + 0] += momh[0].x;
                            cmom[p + 1] += momh[0].y;

                            for (int q = 1; q < (p + 1) / 2; ++q)
                            //for (int q = (p + 1) / 2 - 1; q>=0; --q)
                            {
                                binCft = cftl[((p + 1) / 2) * order + q]; 
                                cmom[p + 0] += binCft * momh[q].x;
                                cmom[p + 1] += binCft * momh[q].y;
                            }
                            
                            
                        }

                        cm += m;
                    }
                }
                countd[k] = cnt;

                for (int s = 0; s < (order * 2 - 1); ++s)
                    momsd[k * (order * 2 - 1) + s] = cmom[s];

                flag = 1;
            }
        }

        __threadfence();
        __syncthreads();

        if (flag != 0) {

            atomicExch((int*)&massd[k], cm);
            //massd[k] = cm;

            k += inc;
            flag = 0;
        }
    }

    //for (int k = 0; k <= nnodesd; ++k)
    //if ((countd[k] != massd[k]) && (massd[k] >= 0))
    
    //for (int k = 0; k <= nnodesd; ++k)
    //if (massd[k] >= 0)
    //    if (countd[k]!=0 || massd[k]!=1)
    //        if (countd[k] != massd[k])
    //            printf("k = %d, count = %d, massd = %d\n", k, countd[k], massd[k]);

    for (int k = 0; k <= nnodesd; ++k)
        if ((massd[k] < 0) && (countd[k]!=0))
            printf("k = %d, count = %d, massd = %d\n", k, countd[k], massd[k]);
}



/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS4, FACTOR4)
void SortKernel2(int nnodesd, int nbodiesd, volatile int* __restrict sortd, const int* __restrict countd, volatile int* __restrict startd, volatile int* __restrict childd)
{
    register int i, j, k, ch, dec, start, bottom;

    bottom = nnodesd - (nbodiesd - 2); //bottomd;

    //printf("startd[-1] = %d\n", startd[nnodesd]);

    dec = blockDim.x * gridDim.x;
    k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

    // iterate over all cells assigned to thread
    while (k >= bottom) {
        start = startd[k];
        if (start >= 0) {
            j = 0;
            for (i = 0; i < 2; i++) {
                ch = childd[k * 2 + i];
                if (ch >= 0) {
                    if (i != j) {
                        // move children to front (needed later for speed)
                        //printf("sort: k = %d, childd = %d, %d, ch = %d, i = %d, j= %d\n", k, childd[k * 2 + 0], childd[k * 2 + 1], ch, i, j);
                        childd[k * 2 + i] = -1;
                        childd[k * 2 + j] = ch;
                        
                    }
                    j++;
                    if (ch >= nbodiesd) {
                        // child is a cell
                        startd[ch] = start;  // set start ID of child
                        start += countd[ch];  // add #bodies in subtree
                    }
                    else {
                        // child is a body
                        sortd[start] = ch;  // record body in 'sorted' array
                        start++;
                    }
                }
            }
            k -= dec;  // move on to next cell
        }
    }
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/
__global__
__launch_bounds__(THREADS5, FACTOR5)
void ForceCalculationKernel2(int nnodesd, int nbodiesd, \
    int* __restrict errd, real itolsqd, real epssqd, const int* __restrict sortd,
    const int* __restrict childd, const real* __restrict momsd,
    const real2* __restrict posd, volatile real2* __restrict veld, 
    const real2* __restrict Msized)

{
register int i, j, k, n, depth, base, sbase, /*diff,*/ pd, nd;
register real2 p, v, dr;
register real r2;
//register real mom[(order * 2 - 1)];
register const real* mom;

register real2 th;

__shared__ volatile int pos[MAXDEPTH * THREADS5 / WARPSIZE], node[MAXDEPTH * THREADS5 / WARPSIZE];

maxdepthd = 28; ////////!!!!!!!!

if (maxdepthd <= MAXDEPTH)
{
    // figure out first thread in each warp (lane 0)
    base = threadIdx.x / WARPSIZE;
    sbase = base * WARPSIZE;
    j = base * MAXDEPTH;
    //diff = threadIdx.x - sbase;

    __syncthreads();
    __threadfence_block();

    // iterate over all bodies assigned to thread
    for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x)
    {
        i = sortd[k];  // get permuted/sorted index
        //if (i != k)
        //    printf("i = %d, k = %d\n", i, k);


        // cache position info
        p.x = posd[i].x;
        p.y = posd[i].y;

        v.x = 0;
        v.y = 0;

        // initialize iteration stack, i.e., push root node onto stack
        depth = j;
        if (sbase == threadIdx.x)
        {
            pos[j] = 0;
            node[j] = nnodesd * 2;
        }

        do
        {
            // stack is not empty
            pd = pos[depth];
            nd = node[depth];

            while (pd < 2)
            {
                // node on top of stack has more children to process
                n = childd[nd + pd];  // load child pointer
                ++pd;

                if (n >= 0)
                {
                    dr = p - posd[n];

                    mom = momsd + (n * (order * 2 - 1));

                    r2 = (dr.x * dr.x + dr.y * dr.y);   // compute distance squared (plus softening)

                    real sumSide = Msized[n].x + Msized[n].y;

                    if ((n < nbodiesd) || __all_sync(0xffffffff, (sumSide * sumSide + epssqd) * itolsqd < r2))
                        {  // check if all threads agree that cell is far enough away (or is a body)

                            real f = mom[0] / realmax(r2, epssqd);
                            v += f * dr;

                            if (order > 1)
                            {
                                real2 cftr = (r2 ? (1 / r2) : 0) * dr;

                                //s=1:
                                th = multz(cftr, cftr);
                                v += multzA(th, mom[1], mom[2]);

                                for (int s = 3; s < (order * 2 - 1); s += 2)
                                {
                                    th = multz(th, cftr);
#ifdef CALCinFLOAT                                    
                                    if (isinf(th.x) || isinf(th.y))
                                    {
                                        //printf("s = %d\n", s);
                                        break;
                                    }
#endif
                                    v += multzA(th, mom[s], mom[s + 1]);
                                }
                            }
                        }
                        else
                        {
                            // push cell onto stack
                            if (sbase == threadIdx.x)
                            {  // maybe don't push and inc if last child
                                pos[depth] = pd;
                                node[depth] = nd;
                            }
                            depth++;
                            pd = 0;
                            nd = n * 2;
                        }
                    }
                    else
                    {
                        pd = 2;  // early out because all remaining children are also zero
                    }
                }
                depth--;  // done with this level
            } while (depth >= j);

            // update velocity
            veld[i].x = -v.y;
            veld[i].y = v.x;
        }
    }
}


/******************************************************************************/
/*** compute force (direct) ***************************************************/
/******************************************************************************/


__global__
//__launch_bounds__(THREADSD, FACTORD)
void ForceDirectCalculationKernel(int nnodesd, int nbodiesd,
    int* __restrict errd,
    real itolsqd, real epssqd,
    const int* __restrict sortd, const int* __restrict childd,
    const real * __restrict momsd,
    const real2* __restrict posd,
    volatile real2* __restrict veld)
{
    __shared__ real2 shr[BLOCKD];
    __shared__ real shg[BLOCKD];

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    real2 pt;
    pt.x = posd[i].x;
    pt.y = posd[i].y;

    real2 vel;
    vel.x = vel.y = 0;

    real2 dr;
    real dr2, izn;


    //vortices
    for (size_t j = 0; j < nbodiesd; j += BLOCKD)
    {
        shr[threadIdx.x].x = posd[(j + threadIdx.x)].x;
        shr[threadIdx.x].y = posd[(j + threadIdx.x)].y;
        shg[threadIdx.x] = momsd[(j + threadIdx.x)*(order * 2 -1)];//momd[(j + threadIdx.x)].gam;

        __syncthreads();

        for (size_t q = 0; q < BLOCKD; ++q)
        {
            if (j + q < nbodiesd)
            {
                dr.x = pt.x - shr[q].x;
                dr.y = pt.y - shr[q].y;
                dr2 = dr.x * dr.x + dr.y * dr.y;

                izn = shg[q] / realmax(dr2, epssqd);// / CUboundDenom(dr2, eps2); //РЎРіР»Р°Р¶РёРІР°С‚СЊ РЅР°РґРѕ!!!

                vel.x -= dr.y * izn;
                vel.y += dr.x * izn;

            }
        }
        __syncthreads();
    }

    if (i < nbodiesd)
    {
        veld[i].x = vel.x;// * iDPI;
        veld[i].y = vel.y;// * iDPI;
    }
    //*/
}


/******************************************************************************/


void KernelsOptimization()
{
    // set L1/shared memory configuration

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel2), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ClearKernel3), hipFuncCachePreferL1);
    hipGetLastError();  // reset error value

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(ForceDirectCalculationKernel), hipFuncCachePreferEqual); //d
    hipGetLastError();  // reset error value
}


/******************************************************************************/





//////////////////
/// Wrappers
//////////////////



    /******************************************************************************/
    /*** initialize memory ********************************************************/
    /******************************************************************************/

    float cuInitializationKernel(int* __restrict errd)
    {
        //fprintf(stderr, "IKKernel\n");
        
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);

        hipEventRecord(start, 0);
        InitializationKernel<<<1, 1>>> (errd);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
        CudaTest("kernel 0 launch failed");
        
        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }


    /******************************************************************************/
    /*** compute center and radius ************************************************/
    /******************************************************************************/
	float McuBoundingBoxKernel(
		int nbodiesd,
		volatile realPoint* __restrict posd,
		volatile realPoint* __restrict Mposd,
		volatile realPoint* __restrict maxrd, volatile realPoint* __restrict minrd)
	{
		hipEvent_t start, stop;
		float time;

		hipEventCreate(&start);  hipEventCreate(&stop);
		hipEventRecord(start, 0);

		MBoundingBoxKernel<<<blocks * FACTOR1, THREADS1>>> (nbodiesd, (real2*)posd, (real2*)Mposd, (real2*)maxrd, (real2*)minrd);
		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

		CudaTest("Mkernel 1 launch failed");

		hipEventDestroy(start);  hipEventDestroy(stop);
		return time;
	}

	/******************************************************************************/
	/*** Morton codes *************************************************************/
	/******************************************************************************/

    float McuMortonCodesKernel(
        int nbodiesd,
        realPoint* __restrict posd,
        int* __restrict MmortonCodesKeyUnsortd, int* __restrict MmortonCodesIdxUnsortd,
        int* __restrict MmortonCodesKeyd, int* __restrict MmortonCodesIdxd,
        intPair* __restrict Mranged
        )
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = (nbodiesd + 31) / 32;
        dim3 Mthreads = 32;

        MMortonCodesKernel << <Mblocks, Mthreads >> > (nbodiesd, (real2*)posd, MmortonCodesKeyUnsortd, MmortonCodesIdxUnsortd);


        ///RadixSort

        RadixSortFromCUB(
            MmortonCodesKeyUnsortd, MmortonCodesKeyd, \
            MmortonCodesIdxUnsortd, MmortonCodesIdxd, \
            nbodiesd, 0, 2 * codeLength);


        //Заполнение нулевой ячейки (диапазон для корня дерева)
        int totalRange[2] = { 0, nbodiesd - 1 };
        cudaCopyVecToDevice(totalRange, Mranged, 2, sizeof(int));


		hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

		CudaTest("Mkernel 1 launch failed");

		hipEventDestroy(start);  hipEventDestroy(stop);

        

		return time;
	}

    /******************************************************************************/
    /*** Morton Internal nodes build **********************************************/
    /******************************************************************************/

    float McuMortonInternalNodesKernel(
        int nbodiesd,
        int* __restrict MmortonCodesKeyd, 
        int* __restrict Mparentd,
        intPair* __restrict Mchildd,
        intPair* __restrict Mranged
    )
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = ((nbodiesd - 1) + 31) / 32;
        dim3 Mthreads = 32;

        MMortonInternalNodesKernel<<<Mblocks, Mthreads>>> (nbodiesd, MmortonCodesKeyd, Mparentd, (int2*)Mchildd, (int2*)Mranged);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("Mkernel 2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;

    }


    /******************************************************************************/
    /*** Morton Internal nodes geometry calculation *******************************/
    /******************************************************************************/
    float McuMortonInternalCellsGeometryKernel(
        int nbodiesd,
        int* __restrict MmortonCodesKeyd,
        realPoint* __restrict Mposd,
        realPoint* __restrict Msized,
        intPair* __restrict Mranged,
        int* __restrict MlevelUnsortd,
        int* __restrict MlevelSortd,
        int* __restrict MindexUnsortd,
        int* __restrict MindexSortd
    )
    {
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        dim3 Mblocks = ((nbodiesd - 1) + 31) / 32;
        dim3 Mthreads = 32;

		MMortonInternalCellsGeometryKernel << <Mblocks, Mthreads >> > (nbodiesd, MmortonCodesKeyd, (real2*)Mposd, (real2*)Msized, (int2*)Mranged,
            MlevelUnsortd, MindexUnsortd);


        RadixSortFromCUB( \
            MlevelUnsortd, MlevelSortd, \
            MindexUnsortd, MindexSortd, \
            nbodiesd-1, 0, 2 * codeLength);


        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("Mkernel 3 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }



	 	 

    /******************************************************************************/
    /*** build tree ***************************************************************/
    /******************************************************************************/   

    float cuClearKernel23(
        int nnodesd, int nbodiesd,
        volatile int* __restrict startd,
        volatile int* __restrict massd,
        const real* __restrict gamd,
        volatile real* __restrict momsd)
    {
        //fprintf(stderr, "CxKernel\n");
        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        ClearKernel2 << <blocks * 1, 1024 >> > (nnodesd, nbodiesd, startd, massd);
        ClearKernel3 << <blocks * 1, 1024 >> > (nnodesd, nbodiesd, gamd, momsd);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel clear2 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }


    /******************************************************************************/
    /*** compute multipole moments for all the cells ******************************/
    /******************************************************************************/
    float cuSummarizationKernel2(
        const int nnodesd, const int nbodiesd,
        volatile int* __restrict countd, const int* __restrict childd,
        volatile int* __restrict massd,
        volatile real* __restrict momsd,
        volatile realPoint* __restrict posd,
        const int* __restrict cftl)
    {
        //fprintf(stderr, "SKKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        SummarizationKernel2 << <blocks * FACTOR3, THREADS3 >> > (nnodesd, nbodiesd, countd, childd, massd, momsd, (real2*)posd, cftl);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 3 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);

        return time;
    }

    /******************************************************************************/
    /*** sort bodies **************************************************************/
    /******************************************************************************/
    float cuSortKernel2(
        int nnodesd, int nbodiesd,
        volatile int* __restrict sortd, const int* __restrict countd,
        volatile int* __restrict startd, volatile int* __restrict childd)
    {
        //fprintf(stderr, "SRKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        SortKernel2 << <blocks * FACTOR4, THREADS4 >> > (nnodesd, nbodiesd, sortd, countd, startd, childd);

        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 4 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }

    /******************************************************************************/
    /*** compute force ************************************************************/
    /******************************************************************************/
    float cuForceCalculationKernel2(
        int nnodesd, int nbodiesd,
        int* __restrict errd,
        real itolsqd, real epssqd,
        const int* __restrict sortd, const int* __restrict childd,
        const real* __restrict momsd,
        const realPoint* __restrict posd,
        volatile realPoint* __restrict veld,
        volatile realPoint* __restrict Msized)
    {
        //fprintf(stderr, "FCKernel\n");

        hipEvent_t start, stop;
        float time;

        hipEventCreate(&start);  hipEventCreate(&stop);
        hipEventRecord(start, 0);

        ForceCalculationKernel2 << <blocks * FACTOR5, THREADS5 >> > (nnodesd, nbodiesd, errd, itolsqd, epssqd, sortd, childd, momsd, (real2*)posd, (real2*)veld, (real2*)Msized);
        hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);

        CudaTest("kernel 5 launch failed");

        hipEventDestroy(start);  hipEventDestroy(stop);
        return time;
    }

    /******************************************************************************/
    /*** compute force (direct) ***************************************************/
    /******************************************************************************/

    float cuForceDirectCalculationKernel(
        int nnodesd, int nbodiesd,
        int* __restrict errd,
        real itolsqd, real epssqd,
        const int* __restrict sortd, const int* __restrict childd,
        const real * __restrict momsd,
        const realPoint * __restrict posd,
        volatile realPoint* __restrict veld)
    {
        //fprintf(stderr, "DFKernel\n");
        
        hipEvent_t startD, stopD;
        float timeD;

        hipEventCreate(&startD);  hipEventCreate(&stopD);
        hipEventRecord(startD, 0);
        
        ForceDirectCalculationKernel<<<(nbodiesd + BLOCKD - 1) / BLOCKD, BLOCKD>>> (nnodesd, nbodiesd, errd, itolsqd, epssqd, sortd, childd, momsd, (real2*)posd, (real2*)veld);
        hipEventRecord(stopD, 0);  hipEventSynchronize(stopD);  hipEventElapsedTime(&timeD, startD, stopD);
        
        CudaTest("kernel direct launch failed");

        hipEventDestroy(startD);  hipEventDestroy(stopD);
        
        return timeD;
    }

}//namespace BHcu